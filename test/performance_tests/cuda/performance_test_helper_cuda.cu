#include "performance_test_helper_cuda.cuh"
#include "../../../src/cuda/cuda_helper.cuh"

namespace
{
    void setCudaThreadCount( uint32_t threadCount )
    {
        multiCuda::CudaDeviceManager::instance().device().setThreadsPerBlock( threadCount );
    }

    uint32_t getMaximumCudaThreadCount()
    {
        return multiCuda::CudaDeviceManager::instance().device().maximumThreadsPerBlock();
    }
}

namespace Performance_Test
{
    namespace Cuda_Helper
    {
        TimerContainerCuda::TimerContainerCuda()
        {
            multiCuda::cudaCheck( hipEventCreate( &_startEvent ) );
            multiCuda::cudaCheck( hipEventCreate( &_stopEvent  ) );
        }

        TimerContainerCuda::~TimerContainerCuda()
        {
            multiCuda::cudaCheck( hipEventDestroy( _startEvent ) );
            multiCuda::cudaCheck( hipEventDestroy( _stopEvent  ) );
        }

        void TimerContainerCuda::start()
        {
            multiCuda::cudaCheck( hipEventRecord( _startEvent, multiCuda::getCudaStream() ) );
        }

        void TimerContainerCuda::stop()
        {
            multiCuda::cudaCheck( hipEventRecord( _stopEvent, multiCuda::getCudaStream() ) );
            multiCuda::cudaCheck( hipEventSynchronize( _stopEvent ) );

            float time = 0.0f;

            multiCuda::cudaCheck( hipEventElapsedTime( &time, _startEvent, _stopEvent ) );

            push( time );
        }

        std::pair < double, double > runPerformanceTestCuda( performanceFunctionCuda function, uint32_t size, uint32_t threadCountDivider )
        {
            setCudaThreadCount( getMaximumCudaThreadCount() / threadCountDivider );
            TimerContainerCuda timer;
            function(timer, size);
            return timer.mean();
        }

        penguinV::Image uniformImage( uint32_t width, uint32_t height )
        {
            return uniformImage( width, height, randomValue<uint8_t>( 256 ) );
        }

        penguinV::Image uniformImage( uint32_t width, uint32_t height, uint8_t value )
        {
            penguinV::ImageCuda image( width, height );

            image.fill( value );

            penguinV::Image imageOut;
            imageOut.swap( image );

            return imageOut;
        }

        std::vector< penguinV::Image > uniformImages( uint32_t count, uint32_t width, uint32_t height )
        {
            std::vector < penguinV::Image > image( count );

            for( std::vector< penguinV::Image >::iterator im = image.begin(); im != image.end(); ++im )
                *im = uniformImage( width, height );

            return image;
        }
    }
}
