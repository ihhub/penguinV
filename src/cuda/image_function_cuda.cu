#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include "image_function_cuda.cuh"
#include "../parameter_validation.h"
#include "../image_function_helper.h"
#include "cuda_types.cuh"
#include "cuda_helper.cuh"

namespace
{
    struct FunctionRegistrator
    {
        Image_Function_Helper::FunctionTableHolder table;

        FunctionRegistrator()
        {
            table.AbsoluteDifference = &Image_Function_Cuda::AbsoluteDifference;
            table.BitwiseAnd         = &Image_Function_Cuda::BitwiseAnd;
            table.BitwiseOr          = &Image_Function_Cuda::BitwiseOr;
            table.BitwiseXor         = &Image_Function_Cuda::BitwiseXor;
            table.ConvertToGrayScale = &Image_Function_Cuda::ConvertToGrayScale;
            table.ConvertToRgb       = &Image_Function_Cuda::ConvertToRgb;
            table.Copy               = &Image_Function_Cuda::Copy;
            table.ExtractChannel     = &Image_Function_Cuda::ExtractChannel;
            table.Fill               = &Image_Function_Cuda::Fill;
            table.GammaCorrection    = &Image_Function_Cuda::GammaCorrection;
            table.Histogram          = &Image_Function_Cuda::Histogram;
            table.Invert             = &Image_Function_Cuda::Invert;
            table.LookupTable        = &Image_Function_Cuda::LookupTable;
            table.SetPixel           = &Image_Function_Cuda::SetPixel;
            table.Maximum            = &Image_Function_Cuda::Maximum;
            table.Minimum            = &Image_Function_Cuda::Minimum;
            table.Subtract           = &Image_Function_Cuda::Subtract;
            table.Threshold          = &Image_Function_Cuda::Threshold;
            table.Threshold2         = &Image_Function_Cuda::Threshold;

            ImageTypeManager::instance().setFunctionTable( PenguinV_Image::ImageCuda().type(), table );
            ImageTypeManager::instance().setConvertFunction( Image_Function_Cuda::ConvertToCuda, PenguinV_Image::Image(), PenguinV_Image::ImageCuda() );
            ImageTypeManager::instance().setConvertFunction( Image_Function_Cuda::ConvertFromCuda, PenguinV_Image::ImageCuda(), PenguinV_Image::Image() );
        }
    };

    const FunctionRegistrator functionRegistrator;

    // The list of CUDA device functions on device side
    __global__ void absoluteDifferenceCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                            uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
            const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
            uint8_t * outX = out + y * rowSizeOut + x;
            (*outX) = ((*in1X) > ( *in2X )) ? ((*in1X) - (*in2X)) : ((*in2X) - (*in1X));
        }
    }

    __global__ void bitwiseAndCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                    uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t idIn1 = y * rowSizeIn1 + x;
            const uint32_t idIn2 = y * rowSizeIn2 + x;
            const uint32_t idOut = y * rowSizeOut + x;
            out[idOut] = in1[idIn1] & in2[idIn2];
        }
    }

    __global__ void bitwiseOrCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                   uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t idIn1 = y * rowSizeIn1 + x;
            const uint32_t idIn2 = y * rowSizeIn2 + x;
            const uint32_t idOut = y * rowSizeOut + x;
            out[idOut] = in1[idIn1] | in2[idIn2];
        }
    }

    __global__ void bitwiseXorCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                    uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t idIn1 = y * rowSizeIn1 + x;
            const uint32_t idIn2 = y * rowSizeIn2 + x;
            const uint32_t idOut = y * rowSizeOut + x;
            out[idOut] = in1[idIn1] ^ in2[idIn2];
        }
    }

    __global__ void convertToGrayScaleCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t colorCount, uint8_t * out, uint32_t rowSizeOut,
                                            uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * data = in + y * rowSizeIn + x * colorCount;
            const uint8_t * dataEnd = data + colorCount;

            uint32_t sum = 0;
            for ( ; data != dataEnd; ++data )
            {
                sum += (*data);
            }

            const uint32_t id = y * rowSizeOut + x;
            out[id] = static_cast<uint8_t>(sum / colorCount);
        }
    }

    __global__ void convertToRgbCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint8_t colorCount,
                                      uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * dataIn = in + y * rowSizeIn + x;

            uint8_t * dataOut = out + y * rowSizeOut + x * colorCount;
            const uint8_t * dataOutEnd = dataOut + colorCount;

            for ( ; dataOut != dataOutEnd; ++dataOut )
            {
                (*dataOut) = (*dataIn);
            }
        }
    }

    __global__ void copyCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            out[y * rowSizeOut + x] = in[y * rowSizeIn + x];
        }
    }

    __global__ void extractChannelCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t colorCount, uint8_t * out, uint32_t rowSizeOut,
                                        uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height )
            out[y * rowSizeOut + x] = in[y * rowSizeIn + x * colorCount];
    }

    __global__ void fillCuda( uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint8_t value )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height )
            data[y * rowSize + x] = value;
    }

    __global__ void flipCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height,
                              bool horizontal, bool vertical )
    {
        const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

        if ( inX < width && inY < height ) {
            const uint32_t outX = horizontal ? (width  - 1 - inX) : inX;
            const uint32_t outY = vertical   ? (height - 1 - inY) : inY;

            out[outY * rowSizeOut + outX] = in[inY * rowSizeIn + inX];
        }
    }

    __global__ void histogramCuda( const uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * histogram )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t id = y * rowSize + x;
            atomicAdd( &histogram[data[id]], 1 );
        }
    }

    __global__ void invertCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            out[y * rowSizeOut + x] = ~in[y * rowSizeIn + x];
        }
    }

    __global__ void isEqualCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t * isEqual )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t partsEqual = static_cast<uint32_t>( in1[y * rowSizeIn1 + x] == in2[y * rowSizeIn2 + x] );
            atomicAnd( isEqual, partsEqual );
        }
    }

    __global__ void lookupTableCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut,
                                     uint32_t width, uint32_t height, uint8_t * table )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            out[y * rowSizeOut + x] = table[in[y * rowSizeIn + x]];
        }
    }

    __global__ void maximumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                 uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
            const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
            uint8_t * outX = out + y * rowSizeOut + x;
            (*outX) = ((*in1X) > ( *in2X )) ? (*in1X) : (*in2X);
        }
    }

    __global__ void minimumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                 uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
            const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
            uint8_t * outX = out + y * rowSizeOut + x;
            (*outX) = ((*in1X) < (*in2X)) ? (*in1X) : (*in2X);
        }
    }

    __global__ void rotateCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut,
                                float inXStart, float inYStart, uint32_t width, uint32_t height, 
                                float cosAngle, float sinAngle )
    {
        uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
        uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

        // Only do something if this thread is for a valid pixel in the output
        if ( outX < width && outY < height ) {
            // Both input coordinates are shifted using the cosAngle, sinAngle, outX, and outY. The shift
            // comes from inverse rotating the horizontal and vertical iterations over the output.

            // Note that inverse rotation by X axis is [cos(angle), -sin(angle)],
            //   and the inverse rotation by Y axis is [sin(angle),  cos(angle)].

            const float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
            const float exactInY = inYStart - sinAngle * outX + cosAngle * outY;

            const int32_t inX = static_cast<int32_t>(exactInX);
            const int32_t inY = static_cast<int32_t>(exactInY);

            // Shift to the output pixel
            out = out + outY * rowSizeOut + outX;

            // Note that we will be taking an average with next pixels, so next pixels need to be in the image too
            if ( inX < 0 || inX >= width - 1 || inY < 0 || inY >= height - 1 ) {
                *out = 0; // We do not actually know what is beyond the image, so set value to 0
            }
            else {
                // Shift to the input pixel
                in = in + inY * rowSizeIn + inX;

                // Now we use a bilinear approximation to find the pixel intensity value. That is, we take an
                // average of pixels (inX, inY), (inX + 1, inY), (inX, inY + 1), and (inX + 1, inY + 1).
                // We add an offset of 0.5 so that conversion to integer is done using rounding.
                const float probX = exactInX - inX;
                const float probY = exactInY - inY;
                const float mean = *in * (1 - probX) * (1 - probY) +
                                   *(in + 1) * probX * (1 - probY) +
                                   *(in + rowSizeIn) * (1 - probX) * probY +
                                   *(in + rowSizeIn + 1) * probX * probY +
                                   0.5f;

                *out = static_cast<uint8_t>(mean);
            }
        }
    }

    __global__ void setPixelCuda( uint8_t * in, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t x, uint32_t y, uint8_t value )
    {
        if ( x < width && y < height ) {
            in[y * rowSize + x] = value;
        }
    }

    __global__ void setPixelCuda( uint8_t * in, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * pointX, uint32_t * pointY, uint32_t pointSize, uint32_t value )
    {
        const uint32_t idPoint = blockIdx.x * blockDim.x + threadIdx.x;

        if ( idPoint < pointSize) {
            const uint32_t x = pointX[idPoint];
            const uint32_t y = pointY[idPoint];
            if ( x < width && y < height ) {
                in[y * rowSize + x] = value;
            }
        }
    }

    __global__ void subtractCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2,
                                  uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint8_t * in1X = in1 + y * rowSizeIn1 + x;
            const uint8_t * in2X = in2 + y * rowSizeIn2 + x;
            uint8_t * outX = out + y * rowSizeOut + x;
            (*outX) = ((*in1X) > ( *in2X )) ? ((*in1X) - (*in2X)) : 0;
        }
    }

    __global__ void thresholdCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height,
                                   uint8_t threshold )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            out[y * rowSizeOut + x] = (in[y * rowSizeIn + x] < threshold) ? 0 : 255;
        }
    }

    __global__ void thresholdCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height,
                                   uint8_t minThreshold, uint8_t maxThreshold )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if ( x < width && y < height ) {
            const uint32_t idIn = y * rowSizeIn + x;
            out[y * rowSizeOut + x] = ((in[idIn] < minThreshold) || (in[idIn] > maxThreshold)) ? 0 : 255;
        }
    }
}

namespace Image_Function_Cuda
{
    Image AbsoluteDifference( const Image & in1, const Image & in2 )
    {
        return Image_Function_Helper::AbsoluteDifference( AbsoluteDifference, in1, in2 );
    }

    void AbsoluteDifference( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function_Helper::AbsoluteDifference( AbsoluteDifference, in1, in2, out );
    }

    Image AbsoluteDifference( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                              uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::AbsoluteDifference( AbsoluteDifference, in1, startX1, startY1, in2, startX2, startY2, width, height );
    }

    void AbsoluteDifference( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                             Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( absoluteDifferenceCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image BitwiseAnd( const Image & in1, const Image & in2 )
    {
        return Image_Function_Helper::BitwiseAnd( BitwiseAnd, in1, in2 );
    }

    void BitwiseAnd( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function_Helper::BitwiseAnd( BitwiseAnd, in1, in2, out );
    }

    Image BitwiseAnd( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                      uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::BitwiseAnd( BitwiseAnd, in1, startX1, startY1, in2, startX2, startY2, width, height );
    }

    void BitwiseAnd( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                     Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( bitwiseAndCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image BitwiseOr( const Image & in1, const Image & in2 )
    {
        return Image_Function_Helper::BitwiseOr( BitwiseOr, in1, in2 );
    }

    void BitwiseOr( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function_Helper::BitwiseOr( BitwiseOr, in1, in2, out );
    }

    Image BitwiseOr( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                     uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::BitwiseOr( BitwiseOr, in1, startX1, startY1, in2, startX2, startY2, width, height );
    }

    void BitwiseOr( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                    Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( bitwiseOrCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image BitwiseXor( const Image & in1, const Image & in2 )
    {
        return Image_Function_Helper::BitwiseXor( BitwiseXor, in1, in2 );
    }

    void BitwiseXor( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function_Helper::BitwiseXor( BitwiseXor, in1, in2, out );
    }

    Image BitwiseXor( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                      uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::BitwiseXor( BitwiseXor, in1, startX1, startY1, in2, startX2, startY2, width, height );
    }

    void BitwiseXor( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                     Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( bitwiseXorCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image ConvertToCuda( const Image & in )
    {
        Image out = ImageCuda().generate( in.width(), in.height(), in.colorCount() );

        ConvertToCuda( in, out );

        return out;
    }

    void ConvertToCuda( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in );
        Image_Function::ParameterValidation( out );

        if ( in.width() != out.width() || in.height() != out.height() ||
             in.colorCount() != out.colorCount() )
            throw imageException( "Bad input parameters in image function" );

        if ( in.alignment() == 1u || (in.rowSize() == in.width() * in.colorCount()) )
        {
            const uint32_t size = in.rowSize() * in.height();

            if ( !multiCuda::cudaSafeCheck( hipMemcpy( out.data(), in.data(), size * sizeof( uint8_t ), hipMemcpyHostToDevice ) ) )
                throw imageException( "Cannot copy a memory to CUDA device" );
        }
        else
        {
            if ( !multiCuda::cudaSafeCheck( hipMemcpy2D( out.data(), out.rowSize(), in.data(), in.rowSize(),
                                                          in.colorCount() * in.width(), in.height(), hipMemcpyHostToDevice ) ) )
                throw imageException( "Cannot copy a memory to CUDA device" );
        }
    }

    Image ConvertFromCuda( const Image & in )
    {
        Image out( in.width(), in.height(), in.colorCount(), 1u );

        ConvertFromCuda( in, out );

        return out;
    }

    void ConvertFromCuda(const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in );
        Image_Function::ParameterValidation( out );

        if ( in.width() != out.width() || in.height() != out.height() ||
             in.colorCount() != out.colorCount() )
            throw imageException( "Bad input parameters in image function" );

        if ( out.alignment() == 1u || (out.rowSize() == out.width() * out.colorCount()) )
        {
            const uint32_t size = in.rowSize() * in.height();

            if ( !multiCuda::cudaSafeCheck( hipMemcpy( out.data(), in.data(), size, hipMemcpyDeviceToHost ) ) )
                throw imageException( "Cannot copy a memory from CUDA device" );
        }
        else
        {
            if ( !multiCuda::cudaSafeCheck( hipMemcpy2D( out.data(), out.rowSize(), in.data(), in.rowSize(),
                                                          in.colorCount() * in.width(), in.height(), hipMemcpyDeviceToHost ) ) )
                throw imageException( "Cannot copy a memory to CUDA device" );
        }
    }

    Image ConvertToGrayScale( const Image & in )
    {
        return Image_Function_Helper::ConvertToGrayScale( ConvertToGrayScale, in );
    }

    void ConvertToGrayScale( const Image & in, Image & out )
    {
        Image_Function_Helper::ConvertToGrayScale( ConvertToGrayScale, in, out );
    }

    Image ConvertToGrayScale( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::ConvertToGrayScale( ConvertToGrayScale, in, startXIn, startYIn, width, height );
    }

    void ConvertToGrayScale( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                             uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( out );

        if ( in.colorCount() == PenguinV_Image::GRAY_SCALE ) {
            Copy( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
            return;
        }

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t colorCount = in.colorCount();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        launchKernel2D( convertToGrayScaleCuda, width, height,
                        inY, rowSizeIn, colorCount, outY, rowSizeOut, width, height );
    }

    Image ConvertToRgb( const Image & in )
    {
        return Image_Function_Helper::ConvertToRgb( ConvertToRgb, in );
    }

    void ConvertToRgb( const Image & in, Image & out )
    {
        Image_Function_Helper::ConvertToRgb( ConvertToRgb, in, out );
    }

    Image ConvertToRgb( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::ConvertToRgb( ConvertToRgb, in, startXIn, startYIn, width, height );
    }

    void ConvertToRgb( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                       uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyRGBImage     ( out );

        if ( in.colorCount() == PenguinV_Image::RGB ) {
            Copy( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
            return;
        }

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t colorCount = out.colorCount();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( convertToRgbCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, colorCount, width, height );
    }

    void Copy( const Image & in, Image & out )
    {
        Image_Function::ParameterValidation( in, out );

        out = in;
    }

    Image Copy( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::Copy( Copy, in, startXIn, startYIn, width, height );
    }

    void Copy( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
               uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );

        const uint8_t colorCount  = Image_Function::CommonColorCount( in, out );
        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        width = width * colorCount;

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn  * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( copyCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height );
    }

    Image ExtractChannel( const Image & in, uint8_t channelId )
    {
        return Image_Function_Helper::ExtractChannel( ExtractChannel, in, channelId );
    }

    void ExtractChannel( const Image & in, Image & out, uint8_t channelId )
    {
        Image_Function_Helper::ExtractChannel( ExtractChannel, in, out, channelId );
    }

    Image ExtractChannel( const Image & in, uint32_t x, uint32_t y, uint32_t width, uint32_t height, uint8_t channelId )
    {
        return Image_Function_Helper::ExtractChannel( ExtractChannel, in, x, y, width, height, channelId );
    }

    void ExtractChannel( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut,
                         uint32_t startYOut, uint32_t width, uint32_t height, uint8_t channelId )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( out );

        if ( channelId >= in.colorCount() )
            throw imageException( "Channel ID for color image is greater than channel count in input image" );

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t colorCount = in.colorCount();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn * colorCount + channelId;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        launchKernel2D( extractChannelCuda, width, height,
                        inY, rowSizeIn, colorCount, outY, rowSizeOut, width, height );
    }

    void Fill( Image & image, uint8_t value )
    {
        image.fill( value );
    }

    void Fill( Image & image, uint32_t x, uint32_t y, uint32_t width, uint32_t height, uint8_t value )
    {
        Image_Function::ParameterValidation( image, x, y, width, height );
        Image_Function::VerifyGrayScaleImage( image );

        const uint32_t rowSize = image.rowSize();

        uint8_t * imageY = image.data() + y * rowSize + x;

        launchKernel2D( fillCuda, width, height,
                        imageY, rowSize, width, height, value );
    }

    Image Flip( const Image & in, bool horizontal, bool vertical )
    {
        return Image_Function_Helper::Flip( Flip, in, horizontal, vertical );
    }

    void Flip( const Image & in, Image & out, bool horizontal, bool vertical )
    {
        Image_Function_Helper::Flip( Flip, in, out, horizontal, vertical );
    }

    Image Flip( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height,
                bool horizontal, bool vertical )
    {
        return Image_Function_Helper::Flip( Flip, in, startXIn, startYIn, width, height, horizontal, vertical );
    }

    void Flip( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
               uint32_t width, uint32_t height, bool horizontal, bool vertical )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( in, out );

        if ( !horizontal && !vertical ) {
            Copy( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        }
        else {
            const uint8_t colorCount = in.colorCount();
            width = width * colorCount;

            const uint32_t rowSizeIn  = in.rowSize();
            const uint32_t rowSizeOut = out.rowSize();

            const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn  * colorCount;
            uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;
            launchKernel2D( flipCuda, width, height,
                            inY, rowSizeIn, outY, rowSizeOut, width, height, horizontal, vertical );
        }
    }

    Image GammaCorrection( const Image & in, double a, double gamma )
    {
        return Image_Function_Helper::GammaCorrection( GammaCorrection, in, a, gamma );
    }

    void GammaCorrection( const Image & in, Image & out, double a, double gamma )
    {
        Image_Function_Helper::GammaCorrection( GammaCorrection, in, out, a, gamma );
    }

    Image GammaCorrection( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height, double a, double gamma )
    {
        return Image_Function_Helper::GammaCorrection( GammaCorrection, in, startXIn, startYIn, width, height, a, gamma );
    }

    void GammaCorrection( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                          uint32_t width, uint32_t height, double a, double gamma )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( in, out );

        const std::vector<uint8_t> & value = Image_Function_Helper::GetGammaCorrectionLookupTable( a, gamma );

        LookupTable( in, startXIn, startYIn, out, startXOut, startYOut, width, height, value );
    }

    uint8_t GetThreshold( const std::vector < uint32_t > & histogram )
    {
        return Image_Function_Helper::GetThreshold( histogram );
    }

    std::vector < uint32_t > Histogram( const Image & image )
    {
        return Image_Function_Helper::Histogram( Histogram, image );
    }

    void Histogram( const Image & image, std::vector < uint32_t > & histogram )
    {
        Image_Function_Helper::Histogram( Histogram, image, histogram );
    }

    std::vector < uint32_t > Histogram( const Image & image, uint32_t x, uint32_t y, uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::Histogram( Histogram, image, x, y, width, height );
    }

    void Histogram( const Image & image, uint32_t x, uint32_t y, uint32_t width, uint32_t height, std::vector < uint32_t > & histogram )
    {
        Image_Function::ParameterValidation( image, x, y, width, height );
        Image_Function::VerifyGrayScaleImage( image );

        histogram.resize( 256u );
        std::fill( histogram.begin(), histogram.end(), 0u );

        const uint32_t rowSize = image.rowSize();

        const uint8_t * imageY = image.data() + y * rowSize + x;

        multiCuda::Array< uint32_t > tableCuda( histogram );

        launchKernel2D( histogramCuda, width, height,
                        imageY, rowSize, width, height, tableCuda.data() );

        histogram = tableCuda.get();
    }

    Image Invert( const Image & in )
    {
        return Image_Function_Helper::Invert( Invert, in );
    }

    void Invert( const Image & in, Image & out )
    {
        Image_Function_Helper::Invert( Invert, in, out );
    }

    Image Invert( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::Invert( Invert, in, startXIn, startYIn, width, height );
    }

    void Invert( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                 uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in, out );
        width = width * colorCount;

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn  * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( invertCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height );
    }

    bool IsEqual( const Image & in1, const Image & in2 )
    {
        Image_Function::ParameterValidation( in1, in2 );

        return IsEqual( in1, 0, 0, in2, 0, 0, in1.width(), in1.height() );
    }

    bool IsEqual( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                  uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2 );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();

        const uint8_t * in1Y = in1.data() + startY1 * rowSizeIn1 + startX1 * colorCount;
        const uint8_t * in2Y = in2.data() + startY2 * rowSizeIn2 + startX2 * colorCount;

        multiCuda::Type< uint32_t > result( 1 );
        launchKernel2D( isEqualCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, width, height, result.data() );

        return ( result.get() != 0 );
    }

    Image LookupTable( const Image & in, const std::vector < uint8_t > & table )
    {
        return Image_Function_Helper::LookupTable( LookupTable, in, table );
    }

    void LookupTable( const Image & in, Image & out, const std::vector < uint8_t > & table )
    {
        Image_Function_Helper::LookupTable( LookupTable, in, out, table );
    }

    Image LookupTable( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height,
                       const std::vector < uint8_t > & table )
    {
        return Image_Function_Helper::LookupTable( LookupTable, in, startXIn, startYIn, width, height, table );
    }

    void LookupTable( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                      uint32_t width, uint32_t height, const std::vector < uint8_t > & table )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );

        if ( table.size() != 256u )
            throw imageException( "Lookup table size is not equal to 256" );

        const uint8_t colorCount = Image_Function::CommonColorCount( in, out );
        width = width * colorCount;

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn  * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        multiCuda::Array< uint8_t > tableCuda( table );

        launchKernel2D( lookupTableCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height, tableCuda.data() );
    }

    Image Maximum( const Image & in1, const Image & in2 )
    {
        return Image_Function_Helper::Maximum( Maximum, in1, in2 );
    }

    void Maximum( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function_Helper::Maximum( Maximum, in1, in2, out );
    }

    Image Maximum( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                   uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::Maximum( Maximum, in1, startX1, startY1, in2, startX2, startY2, width, height );
    }

    void Maximum( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                  Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( maximumCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image Minimum( const Image & in1, const Image & in2 )
    {
        return Image_Function_Helper::Minimum( Minimum, in1, in2 );
    }

    void Minimum( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function_Helper::Minimum( Minimum, in1, in2, out );
    }

    Image Minimum( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                   uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::Minimum( Minimum, in1, startX1, startY1, in2, startX2, startY2, width, height );
    }

    void Minimum( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                  Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( minimumCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    void Rotate( const Image & in, float centerXIn, float centerYIn, Image & out, float centerXOut, float centerYOut, float angle )
    {
        Image_Function::ParameterValidation( in, out );
        Image_Function::VerifyGrayScaleImage( in, out );

        const float cosAngle = cos( angle );
        const float sinAngle = sin( angle );

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint32_t width  = in.width();
        const uint32_t height = in.height();

        uint8_t const * inMem = in.data();
        uint8_t * outMem = out.data();

        // We iterate over the output array in the usual manner; we iterate over the
        // input using inverse rotation of this shift. Doing so, we start the input
        // iteration at the following positions:
        const float inXStart = -( cosAngle * centerXOut + sinAngle * centerYOut) + centerXIn;
        const float inYStart = -(-sinAngle * centerXOut + cosAngle * centerYOut) + centerYIn;

        launchKernel2D( rotateCuda, width, height,
                        inMem, rowSizeIn, outMem, rowSizeOut,
                        inXStart, inYStart, width, height,
                        cosAngle, sinAngle );
    }

    void SetPixel( Image & image, uint32_t x, uint32_t y, uint8_t value )
    {
        Image_Function::ParameterValidation( image );

        if ( x >= image.width() || y >= image.height() )
            throw imageException( "Bad input parameters in image function" );

        launchKernel1D( setPixelCuda, 1,
                        image.data(), image.rowSize(), image.width(), image.height(), x, y, value );
    }

    void SetPixel( Image & image, const std::vector<uint32_t> & X, const std::vector<uint32_t> & Y, uint8_t value )
    {
        Image_Function::ParameterValidation( image );

        if ( X.size() != Y.size() )
            throw imageException( "Bad input parameters in image function" );

        if ( X.size() > 0 ) {
            const uint32_t width = image.width();
            const uint32_t height = image.height();

            for ( size_t i = 0; i < X.size(); ++i ) {
                if ( X[i] >= width || Y[i] >= height )
                    throw imageException( "Bad input parameters in image function" );
            }

            multiCuda::Array<uint32_t> pointX( X );
            multiCuda::Array<uint32_t> pointY( Y );

            launchKernel1D( setPixelCuda, static_cast<uint32_t>( X.size() ),
                            image.data(), image.rowSize(), width, height, pointX.data(), pointY.data(), pointX.size(), value );
        }
    }

    Image Subtract( const Image & in1, const Image & in2 )
    {
        return Image_Function_Helper::Subtract( Subtract, in1, in2 );
    }

    void Subtract( const Image & in1, const Image & in2, Image & out )
    {
        Image_Function_Helper::Subtract( Subtract, in1, in2, out );
    }

    Image Subtract( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                    uint32_t width, uint32_t height )
    {
        return Image_Function_Helper::Subtract( Subtract, in1, startX1, startY1, in2, startX2, startY2, width, height );
    }

    void Subtract( const Image & in1, uint32_t startX1, uint32_t startY1, const Image & in2, uint32_t startX2, uint32_t startY2,
                   Image & out, uint32_t startXOut, uint32_t startYOut, uint32_t width, uint32_t height )
    {
        Image_Function::ParameterValidation( in1, startX1, startY1, in2, startX2, startY2, out, startXOut, startYOut, width, height );

        const uint8_t colorCount = Image_Function::CommonColorCount( in1, in2, out );
        width = width * colorCount;

        const uint32_t rowSizeIn1 = in1.rowSize();
        const uint32_t rowSizeIn2 = in2.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * in1Y = in1.data() + startY1   * rowSizeIn1 + startX1   * colorCount;
        const uint8_t * in2Y = in2.data() + startY2   * rowSizeIn2 + startX2   * colorCount;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut * colorCount;

        launchKernel2D( subtractCuda, width, height,
                        in1Y, rowSizeIn1, in2Y, rowSizeIn2, outY, rowSizeOut, width, height );
    }

    Image Threshold( const Image & in, uint8_t threshold )
    {
        return Image_Function_Helper::Threshold( Threshold, in, threshold );
    }

    void Threshold( const Image & in, Image & out, uint8_t threshold )
    {
        Image_Function_Helper::Threshold( Threshold, in, out, threshold );
    }

    Image Threshold( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height, uint8_t threshold )
    {
        return Image_Function_Helper::Threshold( Threshold, in, startXIn, startYIn, width, height, threshold );
    }

    void Threshold( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                    uint32_t width, uint32_t height, uint8_t threshold )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( in, out );

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        launchKernel2D( thresholdCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height, threshold );
    }

    Image Threshold( const Image & in, uint8_t minThreshold, uint8_t maxThreshold )
    {
        return Image_Function_Helper::Threshold( Threshold, in, minThreshold, maxThreshold );
    }

    void Threshold( const Image & in, Image & out, uint8_t minThreshold, uint8_t maxThreshold )
    {
        Image_Function_Helper::Threshold( Threshold, in, out, minThreshold, maxThreshold );
    }

    Image Threshold( const Image & in, uint32_t startXIn, uint32_t startYIn, uint32_t width, uint32_t height, uint8_t minThreshold,
                     uint8_t maxThreshold )
    {
        return Image_Function_Helper::Threshold( Threshold, in, startXIn, startYIn, width, height, minThreshold, maxThreshold );
    }

    void Threshold( const Image & in, uint32_t startXIn, uint32_t startYIn, Image & out, uint32_t startXOut, uint32_t startYOut,
                    uint32_t width, uint32_t height, uint8_t minThreshold, uint8_t maxThreshold )
    {
        Image_Function::ParameterValidation( in, startXIn, startYIn, out, startXOut, startYOut, width, height );
        Image_Function::VerifyGrayScaleImage( in, out );

        const uint32_t rowSizeIn  = in.rowSize();
        const uint32_t rowSizeOut = out.rowSize();

        const uint8_t * inY  = in.data()  + startYIn  * rowSizeIn  + startXIn;
        uint8_t       * outY = out.data() + startYOut * rowSizeOut + startXOut;

        launchKernel2D( thresholdCuda, width, height,
                        inY, rowSizeIn, outY, rowSizeOut, width, height, minThreshold, maxThreshold );
    }
}
