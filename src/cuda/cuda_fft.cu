#include "hip/hip_runtime.h"
#if !defined(_MSC_VER) && !defined(_WIN32)

#include <hip/hip_runtime.h>
#include "cuda_fft.cuh"
#include "cuda_helper.cuh"
#include "../image_exception.h"

namespace
{
    __global__ void copyFromImageCuda( const uint8_t * in, hipfftComplex * out, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if( x < width && y < height ) {
            const uint32_t id = y * width + x;
            out[id].x = in[id];
            out[id].y = 0;
        }
    }

    __global__ void copyFromFloatCuda( const float * in, hipfftComplex * out, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if( x < width && y < height ) {
            const uint32_t id = y * width + x;
            out[id].x = in[id];
            out[id].y = 0;
        }
    }

    __global__ void copyToImageCuda( const hipfftComplex * in, uint8_t * out, float size, uint32_t width, uint32_t height )
    {
        const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

        if( inX < width && inY < height ) {
            const uint32_t id = inY * width + inX;

            const uint32_t middleX = width / 2;
            const uint32_t middleY = height / 2;

            const uint32_t outX = (inX < middleX) ? middleX + inX : inX - middleX;
            const uint32_t outY = (inY < middleY) ? middleY + inY : inY - middleY;

            out[outY * width + outX] = static_cast<uint8_t>(in[id].x / size + 0.5f);
        }
    }

    __global__ void complexMultiplicationCuda( const hipfftComplex * in1, const hipfftComplex * in2, hipfftComplex * out, uint32_t width, uint32_t height )
    {
        // in1 = A + iB
        // in2 = C + iD
        // out = in1 * (-in2) = (A + iB) * (-C - iD) = - A * C - i(B * C) - i(A * D) + B * D

        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if( x < width && y < height ) {
            const uint32_t id = y * width + x;
            out[id].x = in1[id].x * in2[id].x - in1[id].y * in2[id].y;
            out[id].y = in1[id].x * in2[id].y + in1[id].y * in2[id].x;
        }
    }
}

namespace FFT_Cuda
{
    ComplexData::ComplexData()
    {
    }

    ComplexData::ComplexData( const PenguinV_Image::Image & image )
    {
        set( image );
    }

    ComplexData::ComplexData( const ComplexData & data )
    {
        _copy( data );
    }

    ComplexData::ComplexData( ComplexData && data )
    {
        _swap( data );
    }

    ComplexData::~ComplexData()
    {
        _clean();
    }

    void ComplexData::set( const PenguinV_Image::Image & image )
    {
        if ( image.empty() || image.colorCount() != 1u )
            throw imageException( "Failed to allocate complex data for empty or coloured image" );

        _clean();

        multiCuda::cudaCheck( hipMalloc( &_data, (image.width() * image.height()) * sizeof( hipfftComplex ) ) );

        _width  = image.width();
        _height = image.height();

        launchKernel2D( copyFromImageCuda, _width, _height,
                        image.data(), _data, _width, _height );
    }

    void ComplexData::set( const multiCuda::Array<float> & data )
    {
        if ( data.empty() || _width == 0 || _height == 0 || data.size() != _width * _height )
            throw imageException( "Failed to allocate complex data for empty or coloured image" );

        launchKernel2D( copyFromFloatCuda, _width, _height,
                        data.data(), _data, _width, _height );
    }

    PenguinV_Image::Image ComplexData::get() const
    {
        if ( empty() )
            return PenguinV_Image::Image();

        PenguinV_Image::Image image = PenguinV_Image::ImageCuda().generate( _width, _height );

        const float size = static_cast<float>(image.width() * image.height());

        launchKernel2D( copyToImageCuda, _width, _height,
                        _data, image.data(), size, _width, _height );

        return image;
    }

    void ComplexData::_allocateData( size_t size )
    {
        multiCuda::cudaCheck( hipMalloc( &_data, size ) );
    }

    void ComplexData::_freeData()
    {
        hipFree( _data );
    }

    void ComplexData::_copyData( const BaseComplexData<hipfftComplex> & data )
    {
        if ( !multiCuda::cudaSafeCheck( hipMemcpy( _data, data.data(), _width * _height * sizeof( hipfftComplex ), hipMemcpyDeviceToDevice ) ) )
            throw imageException( "Cannot copy a memory to CUDA device" );
    }

    FFTExecutor::FFTExecutor( uint32_t width_, uint32_t height_ )
        : _plan ( 0 )
    {
        initialize( width_, height_ );
    }

    FFTExecutor::~FFTExecutor()
    {
        _clean();
    }

    void FFTExecutor::directTransform( ComplexData & data )
    {
        directTransform( data, data );
    }

    void FFTExecutor::directTransform( ComplexData & in, ComplexData & out )
    {
        if ( _plan == 0 || !FFT::equalSize( *this, in ) || !FFT::equalSize( in, out ) )
            throw imageException( "Invalid parameters for FFTExecutor" );

        if ( hipfftExecC2C( _plan, in.data(), out.data(), HIPFFT_FORWARD ) != HIPFFT_SUCCESS )
            throw imageException( "Cannot execute direct FFT transform on CUDA device" );
    }

    void FFTExecutor::inverseTransform( ComplexData & data )
    {
        inverseTransform( data, data );
    }

    void FFTExecutor::inverseTransform( ComplexData & in, ComplexData & out )
    {
        if ( _plan == 0 || !FFT::equalSize( *this, in ) || !FFT::equalSize( in, out ) )
            throw imageException( "Invalid parameters for FFTExecutor" );

        if ( hipfftExecC2C( _plan, in.data(), out.data(), HIPFFT_BACKWARD ) != HIPFFT_SUCCESS )
            throw imageException( "Cannot execute inverse FFT transform on CUDA device" );
    }

    void FFTExecutor::complexMultiplication( const ComplexData & in1, ComplexData & in2, ComplexData & out ) const
    {
        if ( !FFT::equalSize( in1, in2 ) || !FFT::equalSize( in1, out ) || in1.width() == 0 || in2.height() == 0 )
            throw imageException( "Invalid parameters for FFTExecutor" );

        launchKernel2D( complexMultiplicationCuda, _width, _height,
                        in1.data(), in2.data(), out.data(), _width, _height );
    }

    void FFTExecutor::_cleanPlans()
    {
        if ( _plan != 0 ) {
            hipfftDestroy( _plan );

            _plan = 0;
        }
    }

    void FFTExecutor::_makePlans()
    {
        if ( hipfftPlan2d( &_plan, _width, _height, HIPFFT_C2C ) != HIPFFT_SUCCESS )
            throw imageException( "Cannot create FFT plan on CUDA device" );
    }
}
#endif
