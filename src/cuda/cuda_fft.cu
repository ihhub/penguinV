#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_fft.cuh"
#include "cuda_helper.cuh"
#include "../image_exception.h"

namespace
{
    __global__ void copyFromImageCuda( const uint8_t * in, hipfftComplex * out, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if( x < width && y < height ) {
            const uint32_t id = y * width + x;
            out[id].x = in[id];
            out[id].y = 0;
        }
    }

    __global__ void copyFromFloatCuda( const float * in, hipfftComplex * out, uint32_t width, uint32_t height )
    {
        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if( x < width && y < height ) {
            const uint32_t id = y * width + x;
            out[id].x = in[id];
            out[id].y = 0;
        }
    }

    __global__ void copyToImageCuda( const hipfftComplex * in, uint8_t * out, float size, uint32_t width, uint32_t height )
    {
        const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

        if( inX < width && inY < height ) {
            const uint32_t id = inY * width + inX;

            const uint32_t middleX = width / 2;
            const uint32_t middleY = height / 2;

            const uint32_t outX = (inX < middleX) ? middleX + inX : inX - middleX;
            const uint32_t outY = (inY < middleY) ? middleY + inY : inY - middleY;

            out[outY * width + outX] = static_cast<uint8_t>(in[id].x / size + 0.5f);
        }
    }

    __global__ void complexMultiplicationCuda( const hipfftComplex * in1, const hipfftComplex * in2, hipfftComplex * out, uint32_t width, uint32_t height )
    {
        // in1 = A + iB
        // in2 = C + iD
        // out = in1 * (-in2) = (A + iB) * (-C - iD) = - A * C - i(B * C) - i(A * D) + B * D

        const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
        const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

        if( x < width && y < height ) {
            const uint32_t id = y * width + x;
            out[id].x = in1[id].x * in2[id].x - in1[id].y * in2[id].y;
            out[id].y = in1[id].x * in2[id].y + in1[id].y * in2[id].x;
        }
    }
}

namespace FFT_Cuda
{
    ComplexData::ComplexData()
        : _data  ( NULL )
        , _width ( 0 )
        , _height( 0 )
    {
    }

    ComplexData::ComplexData( const PenguinV_Image::Image & image )
        : _data  ( NULL )
        , _width ( 0 )
        , _height( 0 )
    {
        set( image );
    }

    ComplexData::ComplexData( const ComplexData & data )
        : _data  ( NULL )
        , _width ( 0 )
        , _height( 0 )
    {
        _copy( data );
    }

    ComplexData::ComplexData( ComplexData && data )
        : _data  ( NULL )
        , _width ( 0 )
        , _height( 0 )
    {
        _swap( data );
    }

    ComplexData & ComplexData::operator=( const ComplexData & data )
    {
        _copy( data );

        return *this;
    }

    ComplexData & ComplexData::operator=( ComplexData && data )
    {
        _swap( data );

        return *this;
    }

    ComplexData::~ComplexData()
    {
        _clean();
    }

    void ComplexData::set( const PenguinV_Image::Image & image )
    {
        if( image.empty() || image.colorCount() != 1u )
            throw imageException( "Failed to allocate complex data for empty or coloured image" );

        _clean();

        multiCuda::cudaCheck( hipMalloc( &_data, (image.width() * image.height()) * sizeof( hipfftComplex ) ) );

        _width  = image.width();
        _height = image.height();

        launchKernel2D( copyFromImageCuda, _width, _height,
                        image.data(), _data, _width, _height );
    }

    void ComplexData::set( const multiCuda::Array<float> & data )
    {
        if( data.empty() || _width == 0 || _height == 0 || data.size() != _width * _height )
            throw imageException( "Failed to allocate complex data for empty or coloured image" );

        launchKernel2D( copyFromFloatCuda, _width, _height,
                        data.data(), _data, _width, _height );
    }

    PenguinV_Image::Image ComplexData::get() const
    {
        if( empty() )
            return PenguinV_Image::Image();

        PenguinV_Image::Image image = PenguinV_Image::ImageCuda().generate( _width, _height );

        const float size = static_cast<float>(image.width() * image.height());

        launchKernel2D( copyToImageCuda, _width, _height,
                        _data, image.data(), size, _width, _height );

        return image;
    }

    void ComplexData::resize( uint32_t width_, uint32_t height_ )
    {
        if( (width_ != _width || height_ != _height) && width_ != 0 && height_ != 0 ) {
            _clean();

            multiCuda::cudaCheck( hipMalloc( &_data, (width_ * height_) * sizeof( hipfftComplex ) ) );

            _width  = width_;
            _height = height_;
        }
    }

    hipfftComplex * ComplexData::data()
    {
        return _data;
    }

    const hipfftComplex * ComplexData::data() const
    {
        return _data;
    }

    uint32_t ComplexData::width() const
    {
        return _width;
    }

    uint32_t ComplexData::height() const
    {
        return _height;
    }

    bool ComplexData::empty() const
    {
        return _data == NULL;
    }

    void ComplexData::_clean()
    {
        if( _data != NULL ) {
            hipFree( _data );
            _data = NULL;
        }

        _width  = 0;
        _height = 0;
    }

    void ComplexData::_copy( const ComplexData & data )
    {
        _clean();

        resize( data._width, data._height );

        if( !empty() ) {
            if( !multiCuda::cudaSafeCheck( hipMemcpy( _data, data._data, _width * _height * sizeof( hipfftComplex ), hipMemcpyDeviceToDevice ) ) )
                throw imageException( "Cannot copy a memory to CUDA device" );
        }
    }

    void ComplexData::_swap( ComplexData & data )
    {
        std::swap( _data, data._data );
        std::swap( _width, data._width );
        std::swap( _height, data._height );
    }

    FFTExecutor::FFTExecutor()
        : _plan  ( 0 )
        , _width ( 0 )
        , _height( 0 )
    {
    }

    FFTExecutor::FFTExecutor( uint32_t width_, uint32_t height_ )
        : _plan  ( 0 )
        , _width ( 0 )
        , _height( 0 )
    {
        initialize( width_, height_ );
    }

    FFTExecutor::~FFTExecutor()
    {
        _clean();
    }

    void FFTExecutor::initialize( uint32_t width_, uint32_t height_ )
    {
        if( width_ == 0 || height_ == 0 )
            throw imageException( "Invalid parameters for FFTExecutor" );

        _clean();

        if( hipfftPlan2d( &_plan, width_, height_, HIPFFT_C2C ) != HIPFFT_SUCCESS )
            throw imageException( "Cannot create FFT plan on CUDA device" );

        _width  = width_;
        _height = height_;
    }

    uint32_t FFTExecutor::width() const
    {
        return _width;
    }

    uint32_t FFTExecutor::height() const
    {
        return _height;
    }

    void FFTExecutor::directTransform( ComplexData & data )
    {
        directTransform( data, data );
    }

    void FFTExecutor::directTransform( ComplexData & in, ComplexData & out )
    {
        if( _plan == 0 || _width != in.width() || _height != in.height() || _width != out.width() || _height != out.height() )
            throw imageException( "Invalid parameters for FFTExecutor" );

        if( hipfftExecC2C( _plan, in.data(), out.data(), HIPFFT_FORWARD ) != HIPFFT_SUCCESS )
            throw imageException( "Cannot execute direct FFT transform on CUDA device" );
    }

    void FFTExecutor::inverseTransform( ComplexData & data )
    {
        inverseTransform( data, data );
    }

    void FFTExecutor::inverseTransform( ComplexData & in, ComplexData & out )
    {
        if( _plan == 0 || _width != in.width() || _height != in.height() || _width != out.width() || _height != out.height() )
            throw imageException( "Invalid parameters for FFTExecutor" );

        if( hipfftExecC2C( _plan, in.data(), out.data(), HIPFFT_BACKWARD ) != HIPFFT_SUCCESS )
            throw imageException( "Cannot execute inverse FFT transform on CUDA device" );
    }

    void FFTExecutor::complexMultiplication( ComplexData & in1, ComplexData & in2, ComplexData & out ) const
    {
        if( in1.width() != in2.width() || in1.height() != in2.height() || in1.width() != out.width() || in1.height() != out.height() ||
            in1.width() == 0 || in1.height() == 0 )
            throw imageException( "Invalid parameters for FFTExecutor" );

        launchKernel2D( complexMultiplicationCuda, _width, _height,
                        in1.data(), in2.data(), out.data(), _width, _height );
    }

    void FFTExecutor::_clean()
    {
        if( _plan != 0 ) {
            hipfftDestroy( _plan );

            _plan = 0;
        }

        _width  = 0;
        _height = 0;
    }
}
